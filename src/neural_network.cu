#include "neural_network.h"
#include "nn_utils/nn_exception.h"

NeuralNetwork::NeuralNetwork(float learning_rate) :
	learning_rate(learning_rate)
{ }

NeuralNetwork::~NeuralNetwork() {
	for (auto layer : layers) {
		delete layer;
	}
}

void NeuralNetwork::addLayer(NNLayer* layer) {
	this->layers.push_back(layer);
}

Matrix NeuralNetwork::forward(Matrix X) {
	Matrix Z = X;

	for (auto layer : layers) {
		Z = layer->forward(Z);
	}

	Y = Z;
	return Y;
}

void NeuralNetwork::backward(Matrix predictions, Matrix target) {
	dY.allocateMemoryIfNotAllocated(predictions.shape);
	Matrix error = func->dCost(predictions, target, dY);

	for (auto it = this->layers.rbegin(); it != this->layers.rend(); it++) {
		error = (*it)->backward(error, learning_rate);
	}

	hipDeviceSynchronize();
}

std::vector<NNLayer*> NeuralNetwork::getLayers() const {
	return layers;
}

void NeuralNetwork::setCostFunction(CostFunction * func) {
	this->func = func;
}

#include "hip/hip_runtime.h"
#include "bce_cost.h"
#include "../nn_utils/nn_exception.h"

#include <math.h>
#include <iostream>
#include <assert.h>

__global__ void binaryCrossEntropyCost(float* predictions, float* target,
									   int size, float* cost) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;

	if (index < size) {
		float partial_cost = target[index] * logf(predictions[index])
				+ (1.0f - target[index]) * logf(1.0f - predictions[index]);
		atomicAdd(cost, - partial_cost / size);
	}
}

__global__ void dBinaryCrossEntropyCost(float* predictions, float* target, float* dY,
								     	int size) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	// dY[index] 存储了损失函数对于第 index 个预测值的梯度。
	if (index < size) dY[index] = -1.0 * ( target[index]/predictions[index] - (1 - target[index])/(1 - predictions[index]) );
}

BCECost::BCECost(std::string name) {
	this->name = name;
}

BCECost::~BCECost() {}

float BCECost::cost(Matrix predictions, Matrix target) {
	assert(predictions.shape.x == target.shape.x);

	float* cost;
	hipMallocManaged(&cost, sizeof(float));
	*cost = 0.0f;

	dim3 block_size(256);
	dim3 num_of_blocks((predictions.shape.x + block_size.x - 1) / block_size.x);
	// std::cout << predictions.shape.x << std::endl;
	binaryCrossEntropyCost<<<num_of_blocks, block_size>>>(predictions.data_device.get(),
														  target.data_device.get(),
														  predictions.shape.x, cost);
	hipDeviceSynchronize();
	NNException::throwIfDeviceErrorsOccurred("Cannot compute binary cross entropy cost.");

	float cost_value = *cost;
	// std::cout << cost_value << std::endl;
	hipFree(cost);

	return cost_value;
}

Matrix BCECost::dCost(Matrix predictions, Matrix target, Matrix dY) {
	assert(predictions.shape.x == target.shape.x);

	dim3 block_size(256);
	dim3 num_of_blocks((predictions.shape.x + block_size.x - 1) / block_size.x);
	dBinaryCrossEntropyCost<<<num_of_blocks, block_size>>>(predictions.data_device.get(),
														   target.data_device.get(),
														   dY.data_device.get(),
														   predictions.shape.x);
	NNException::throwIfDeviceErrorsOccurred("Cannot compute derivative for binary cross entropy.");

	return dY;
}
